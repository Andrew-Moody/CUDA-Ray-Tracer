#include "hip/hip_runtime.h"
#include "raytracing.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <vector>

#include "vec3.cuh"
#include "ray.cuh"
#include "camera.cuh"

#include "hittable.cuh"
#include "sphere.cuh"

#include "scene.cuh"

namespace rtw
{
#define CHECK(result) cuda_assert(result, __FILE__, __LINE__);
	inline void cuda_assert(hipError_t result, const char* file, int line)
	{
		if (result)
		{
			std::cout << hipGetErrorString(result) << "" << file << "" << line << '\n';
		}
	}

	__global__ void renderRayKernel(float* buffer, Camera camera, Scene scene)
	{
		int strideX = blockDim.x * gridDim.x;
		int thread = threadIdx.x + blockIdx.x * blockDim.x;

		for (int pixel = thread; pixel < camera.totalPixels(); pixel += strideX)
		{
			uint32_t pcgState = pixel;

			Vec3 worldPos = camera.getWorldPosFromPixelIndex(pixel);

			Vec3 color{};

			const int nSamples = camera.nSamples();

			for (int i = 0; i < nSamples; ++i)
			{
				// Create a ray from a random point on DOF disk to random point in the pixel
				Ray ray = camera.generateRay(worldPos, pcgState);

				//color += scene.getColor(ray, camera.nBounces(), pcgState);
				
				color += ray.direction();
			}

			//color *= 1.0f / camera.nSamples();

			// Seems to use 14 registers on its own vectorization can sometimes help
			// but might not when register bound
			//// Write color to buffer
			int index = 3 * pixel;
			buffer[index] = color.x();
			buffer[index + 1] = color.y();
			buffer[index + 2] = color.z();
		}
	}

	struct Transform
	{
		float x, y, z, r;
		int index;
	};

	std::vector<float> rtw::renderGPU(const Camera& camera, const Scene& scene)
	{	
		hipDeviceProp_t prop;
		CHECK(hipGetDeviceProperties(&prop, 0));

		// total number of threads that can theoretically be resident
		// 10 mp * 2048 threads per mp
		const int maxThreadsTotal = 10 * 2048;
		//const int maxThreadsPerBlock = 1024;
		//const int maxBlocksTotal = 10 * 32;

		// want to pick multiples of 32 to maximize warp usage
		const int threadsPerBlock{ 256 };
		const int blocks{ maxThreadsTotal / threadsPerBlock };


		// Allocate memory for the frame buffer
		float* buffer{};
		const size_t bufferSize{ static_cast<size_t>(camera.totalPixels() * camera.nChannels()) };
		CHECK(hipMallocManaged(&buffer, bufferSize * sizeof(float)));


		// Copy spheres and colliders to device and wrap in a scene
		Sphere* spheres{};
		Collider* colliders{};
		int numSpheres = scene.getSphereCount();
		CHECK(hipMalloc(&spheres, numSpheres * sizeof(Sphere)));
		CHECK(hipMalloc(&colliders, numSpheres * sizeof(Collider)));
		CHECK(hipMemcpy(spheres, scene.getSpheres(), numSpheres * sizeof(Sphere), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(colliders, scene.getColliders(), numSpheres * sizeof(Collider), hipMemcpyHostToDevice));
		CHECK(hipDeviceSynchronize());
		Scene sceneDevice{ spheres, colliders, numSpheres };


		// Render the frame
		renderRayKernel<<<blocks, threadsPerBlock>>>(buffer, camera, sceneDevice);
		CHECK(hipGetLastError());
		CHECK(hipDeviceSynchronize());

		// Copy data from managed memory to normal memory
		std::vector<float> data{};
		data.assign(buffer, buffer + bufferSize);

		// do this off kernel to remove some register pressure
		for (auto & value : data)
		{
			value /= camera.nSamples();
		}

		// Release device memory
		CHECK(hipFree(spheres));
		CHECK(hipFree(colliders));
		CHECK(hipFree(buffer));

		return data;
	}
}
